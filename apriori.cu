#include "hip/hip_runtime.h"
/************************************************************************
Author - Aman Maldar
Simple code - parallel version of data association.
Static value of minSupport=1. This will show all the pairs generated.
File = 6entries.txt
Limitation - Generates only till set of 4 pairs as of now.
It needs multiple changes for the data structure as well. Need to reconfigure it.

Data: (6entries.txt)
2 3 4
1 2 3 4 5
4 5 6 7
1 2
1 2 3 4
2 3 5 7 8


*************************************************************************/
#include "apriori.h"
#include "functions.h"


double minSupp = 0.001; // 0.001;

void Execute(int argc){

    parse_database(argc);

    L1.push_back(0);    // initialized first index with 0 as we are not using it.
    //minSupport = round(minSupp *  TID_Transactions);
    minSupport = 1;
    // Following code generates single items which have support greater than min_sup
    // compare the occurrence of the object against minSupport

    cout << "\n Support:" << minSupport << endl << "\n";
    //Generate L1 - filtered single items ? I think this should be C1, not L1.

    for (int i=1; i<= maxItemID; i++)
    {
        if(itemIDcount[i] >= minSupport){
            L1.push_back(i);     //push TID into frequentItem
            one_freq_itemset++;
            cout << "1 Frequent Item is: (" << i << ") Freq is: " << itemIDcount[i] << endl;
        }
    }
    cout << "one_freq_itemset:      " << one_freq_itemset << endl << "\n";
    //******************************************************************************************************************
    //Generate L2 .  Make a pair of frequent items in L1
    for (int i=1;i <= L1.size() -1 -1; i++)     //-1 is done for eliminating first entry
    {
        for (int j=i+1;j <= L1.size() -1; j++){
            twoStruct.a = L1[i];
            twoStruct.b = L1[j];
            L2.push_back(twoStruct);
            cout << "2 Items are: (" <<L1[i]<< "," << L1[j] << ") " << endl;

        }
    }
    //******************************************************************************************************************
    //Generate C2. Prune L2 . Compare against min_support and remove less frequent items.
 
    cout << "two_freq_itemset:      " << two_freq_itemset << endl << "\n";

    //******************************************************************************************************************

    
    //work till pair of 2
   // return 0;
    
   
}   // end Execute



int main(int argc, char **argv){

    auto start = chrono::high_resolution_clock::now();

    Execute(argc);

    auto end = chrono::high_resolution_clock::now();
    chrono::duration<double> el = end - start;
    cout<<"Execution time is:     " << el.count() * 1000 << " mS " << endl;

    return 0;
}






