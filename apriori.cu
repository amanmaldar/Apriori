#include "hip/hip_runtime.h"
/************************************************************************
Author - Aman Maldar
Simple code - parallel version of data association.
Static value of minSupport=1. This will show all the pairs generated.
File = 6entries.txt
Limitation - Generates only till set of 4 pairs as of now.
It needs multiple changes for the data structure as well. Need to reconfigure it.

Data: (6entries.txt)
2 3 4
1 2 3 4 5
4 5 6 7
1 2
1 2 3 4
2 3 5 7 8


*************************************************************************/
#include "apriori.hcu"
#include "functions.hcu"


double minSupp = 0.001; // 0.001;
/*
__shared__ int smem[128];

__global__ void prefix_scan_kernel (int *b_d, int *a_d, int n, int depth) {
while (tid < n) {
        smem[threadIdx.x] = a_d[tid];       // each thread copy data to shared memory
        __syncthreads();                    // wait for all threads

        //if (tid%16384 == 0 ) {   smem[tid] += res; __syncthreads();  } // result are written at the end*  

        offset = 1;                 //1->2->4->8
        for (d =0; d < depth ; d++) {                    

            if (threadIdx.x >= offset) {  
                smem[threadIdx.x] += smem[threadIdx.x-offset] ;           //after writing to smem do synchronize
                __syncthreads();      
            } // end if

            offset *=2;
        } // end for loop

        b_d[tid] = smem[threadIdx.x];        // *write the result to array b_d[tid] location
        __syncthreads();                    // wait for all threads to write results
        
        //if ((tid + 1) % 16384 == 0) { inc++; printf("\n incremented %d times\n", inc);}
        tid += 16384;               //there are no actual grid present, we just increment the tid to fetch next elemennts from input array.
        
    } // end while (tid < n)
} // end kernel function

*/
void Execute(int argc){

    	parse_database(argc);
	vector <int> globalMap;     // convert itemId_TidMapping into long array

	int k =0;                   // global pointer for globalMap
	for(int i=1;i<itemId_TidMapping.size();i++){
		for(int j=1;j<itemId_TidMapping.at(i).size();j++){
			vector <int> tmp = itemId_TidMapping[i];
			globalMap.push_back(tmp[j]);
			k++;
		}
		globalMap.push_back(-1);    // seperate mappings by -1
	}
	cout << " Printing itemId_TidMapping as array: " << endl;
	for(int i =0;i<globalMap.size();i++){
		cout << globalMap[i] << " " ;
	}cout << endl;
	
	
	//int numberOfBlocks = 1;
	//int threadsInBlock = 100;
	  
    L1.push_back(0);    // initialized first index with 0 as we are not using it.
    //minSupport = round(minSupp *  TID_Transactions);
    minSupport = 1;
    // Following code generates single items which have support greater than min_sup
    // compare the occurrence of the object against minSupport

    cout << "\n Support:" << minSupport << endl << "\n";
    //Generate L1 - filtered single items ? I think this should be C1, not L1.

    for (int i=1; i<= maxItemID; i++)
    {
        if(itemIDcount[i] >= minSupport){
            L1.push_back(i);     //push TID into frequentItem
            one_freq_itemset++;
            cout << "1 Frequent Item is: (" << i << ") Freq is: " << itemIDcount[i] << endl;
        }
    }
    cout << "one_freq_itemset:      " << one_freq_itemset << endl << "\n";
    //******************************************************************************************************************
    //Generate L2 .  Make a pair of frequent items in L1
    for (int i=1;i <= L1.size() -1 -1; i++)     //-1 is done for eliminating first entry
    {
        for (int j=i+1;j <= L1.size() -1; j++){
            twoStruct.a = L1[i];
            twoStruct.b = L1[j];
            L2.push_back(twoStruct);
            cout << "2 Items are: (" <<L1[i]<< "," << L1[j] << ") " << endl;

        }
    }
    //******************************************************************************************************************
    //Generate C2. Prune L2 . Compare against min_support and remove less frequent items.
 
	vector <vector <int>> *a_d; //device storage pointers
        //hipMalloc ((void **) &a_d, sizeof (itemId_TidMapping));
	//hipMemcpy (a_d, itemId_TidMapping, sizeof (itemId_TidMapping), hipMemcpyHostToDevice);

	//prefix_scan_kernel <<< numberOfBlocks,threadsInBlock >>> (a_d,8);

 
    cout << "two_freq_itemset:      " << two_freq_itemset << endl << "\n";

    //******************************************************************************************************************

    
    //work till pair of 2
    return;
    
   
}   // end Execute



int main(int argc, char **argv){

    auto start = chrono::high_resolution_clock::now();

    Execute(argc);

    auto end = chrono::high_resolution_clock::now();
    chrono::duration<double> el = end - start;
    cout<<"Execution time is:     " << el.count() * 1000 << " mS " << endl;

    return 0;
}






